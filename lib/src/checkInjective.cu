
#include <hip/hip_runtime.h>
#include <random>
#include <stdio.h>



int getRandom(int n) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, n);
    return dis(gen);
}

__global__
void check(int *fDictX, int *fDictY, int sizeOfFDict, int currInjective) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < sizeOfFDict) {
        for (int i=idx; i < sizeOfFDict && currInjective; i++) {
            if (fDictY[idx] == fDictY[i] && fDictX[idx] != fDictX[i]) {
                currInjective = 0;
            }
        }
    }
}


int injective(int maxDomain, int maxCodomain) {
    int INJECTIVE = 1;
    // get a device copy of INJECTIVE
    //int d_INJECTIVE;
    //cudaMalloc(&d_INJECTIVE, sizeof(int));
    //cudaMemcpy(d_INJECTIVE, INJECTIVE, sizeof(int), cudaMemcpyHostToDevice);
    // ****

    int *domainX, *domainY, *d_domainX, *d_domainY;  // host, device
    domainX = (int*)malloc(maxDomain * sizeof(int));
    domainY = (int*)malloc(maxDomain * sizeof(int));

    hipMalloc(&d_domainX, maxDomain*sizeof(int));
    hipMalloc(&d_domainY, maxDomain*sizeof(int));

    for (int i = 0; i < maxDomain; i++) {
        domainX[i] = i;
        domainY[i] = getRandom(maxCodomain);
    }

    hipMemcpy(d_domainX, domainX, maxDomain*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_domainY, domainY, maxDomain*sizeof(int), hipMemcpyHostToDevice);

    check<<<(maxDomain+255)/256, 256>>>(d_domainX, d_domainY, maxDomain, INJECTIVE);

    //cudaMemcpy(INJECTIVE, d_INJECTIVE, sizeof(int), cudaMemcpyDeviceToHost);
    
    hipFree(d_domainX);
    hipFree(d_domainY);
    free(domainX);
    free(domainY);

    return INJECTIVE;

}
    

int main() {
    int test = injective(2560, 10240);
    printf("%s\n", test);
    
    return 0;
}

    


    

