
#include <hip/hip_runtime.h>
#include <random>



int getRandom(int n) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, n);
    return dis(gen);
}

__global__
void check(int *fDictX, int *fDictY, int sizeOfFDict, int *currInjective) {
    int idxInit = blockIdx.x * blockDim.x + threadIdx.x;  // this is the initial global index

    if (idxInit < sizeOfFDict) {  // make sure we're not out of bounds
        // grid-stride loop 
        for (int idx = idxInit; idx <= sizeOfFDict; idx += blockDim.x * gridDim.x) {
            // this inner loop will do the work for a thread in each grid
            for (int i=idx; i < sizeOfFDict; i++) {
                if (fDictY[idx] == fDictY[i] && fDictX[idx] != fDictX[i]) {
                    currInjective[idx] = 0;  // this will insert a zero into INJECTIVE
                    break;
                }
            }
        }
    }
}


extern "C" {
    int injective(int maxDomain, int maxCodomain) {

        // declare the arrays
        int *domainX, *domainY, *INJECTIVE;

        // allocated Unified Memory
        hipMallocManaged(&domainX, maxDomain*sizeof(int));
        hipMallocManaged(&domainY, maxDomain*sizeof(int));
        hipMallocManaged(&INJECTIVE, maxDomain*sizeof(int));


        // initialize function
        for (int i = 0; i < maxDomain; i++) {
            domainX[i] = i;
            domainY[i] = getRandom(maxCodomain);
            INJECTIVE[i] = 1;  // here we assume that the function is injective
        }


        // actually make the kernel call
        // this is currently running with thread blocks of size 256. I may find a better number
        // to tune that too. It's <<< numBlocks, numThreads >>>
        check<<<(maxDomain+255)/256, 256>>>(domainX, domainY, maxDomain, INJECTIVE);
        hipDeviceSynchronize();  // sync the host and device

        // error checking
        if (hipSuccess != hipGetLastError()) {
            hipFree(domainX);
            hipFree(domainY);
            hipFree(INJECTIVE);
            return -1;  // this will be used to raise a RuntimeError 
        }

        // free the unified memory
        hipFree(domainX);
        hipFree(domainY);

        for (int i = 0; i < maxDomain; i++) {
            if (INJECTIVE[i] == 0) {  // 0 => false so the function is not injective
                hipFree(INJECTIVE);
                return 0;
            }
        }

        hipFree(INJECTIVE);
        return 1;  // the function is injective

    }
}
    
    
